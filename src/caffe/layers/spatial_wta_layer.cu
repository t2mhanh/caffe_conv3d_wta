#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/spatial_wta_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SpatialWtaForward(const int nthreads,
    const Dtype* const bottom_data, const int num, const int channels,
    const int height, const int width, const int wta_height,
    const int wta_width, const int kernel_h, const int kernel_w,
    const int stride_h, const int stride_w,
    Dtype* const top_data, int* mask, Dtype* top_mask) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int ww = index % wta_width;
    const int wh = (index / wta_width) % wta_height;
    const int c = (index / wta_width / wta_height) % channels;
    const int n = index / wta_width / wta_height / channels;

    int hstart_lower = static_cast<int>(floor(static_cast<float>(
        wh - kernel_h)/stride_h) + 1)*stride_h;
    int wstart_lower = static_cast<int>(floor(static_cast<float>(
        ww - kernel_w)/stride_w) + 1)*stride_w;
    hstart_lower = max(hstart_lower,0);
    wstart_lower = max(wstart_lower,0);
    //int hstart_lower = max(ceil((wh - kernel_h)/stride_h)*stride_h,0);
    // const int hstart_lower = max(((wh - kernel_h)/stride_h+1)*stride_h,0);
    // int hstart_upper = wh;
    int hstart_upper = min(wh,wta_height - kernel_h);
    //const int wstart_lower = max(ceil((ww - kernel_w)/stride_w),0);
    // const int wstart_lower = max(((ww - kernel_w)/stride_w+1)*stride_w,0);
    // int wstart_upper = ww;
    int wstart_upper = min(ww,wta_width - kernel_w);
    for (int hstart = hstart_lower; hstart <= hstart_upper; hstart += stride_h){
      for (int wstart = wstart_lower; wstart <= wstart_upper; wstart += stride_w){
        const int hend = min(hstart + kernel_h, height);
        const int wend = min(wstart + kernel_w, width);
        Dtype maxval = -FLT_MAX;
        int maxidx = -1;
        const Dtype* const bottom_slice =
            bottom_data + (n * channels + c) * height * width;
        for (int h = hstart; h < hend; ++h) {
          for (int w = wstart; w < wend; ++w) {
            // if (bottom_slice[h * width + w] > maxval) {
              if (fabs(bottom_slice[h * width + w]) > maxval) {
                maxidx = h * width + w;
                maxval = fabs(bottom_slice[maxidx]);
              // maxval = bottom_slice[maxidx];
            }
          }
        }
        maxidx += (n * channels + c) * height * width;
        if (index == maxidx){
          top_data[index] = bottom_data[index];
          if (mask) {
            mask[index] = 1;
          } else {
            top_mask[index] = 1;
          }
        }
        // -- these lines may set previous maximum values to 0
        // else {
        //   top_data[index] = 0;
        //   if (mask) {
        //     mask[index] = 0;
        //   } else {
        //     top_mask[index] = 0;
        //   }
        // }
      }
    }
  }
}

template <typename Dtype>
void SpatialWtaLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
       const vector<Blob<Dtype>*>& top) {
const Dtype* bottom_data = bottom[0]->gpu_data();
Dtype* top_data = top[0]->mutable_gpu_data();
int count = top[0]->count();
caffe_gpu_set(count, Dtype(0.), top_data);
// We'll output the mask to top[1] if it's of size >1.
const bool use_top_mask = top.size() > 1;
int* mask = NULL;
Dtype* top_mask = NULL;
if (use_top_mask) {
    top_mask = top[1]->mutable_gpu_data();
    caffe_gpu_set(count, Dtype(0.), top_mask);
} else {
    mask = max_mask_.mutable_gpu_data();
    caffe_gpu_set(count, 0 , mask);
}
  // NOLINT_NEXT_LINE(whitespace/operators)
  SpatialWtaForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, bottom[0]->num(), channels_,
      height_, width_, wta_height_, wta_width_, kernel_h_,
      kernel_w_, stride_h_, stride_w_, top_data,
      mask, top_mask);
CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__global__ void SpatialWtaBackward(const int nthreads, const Dtype* const top_diff,
    const int* const mask, const Dtype* const top_mask, Dtype* const bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    if (mask) {
      bottom_diff[index] = top_diff[index] * mask[index];
    } else {
      bottom_diff[index] = top_diff[index] * top_mask[index];
          }
        }
      }

template <typename Dtype>
void SpatialWtaLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    // We'll output the mask to top[1] if it's of size >1.
    const bool use_top_mask = top.size() > 1;
    const int* mask = NULL;
    const Dtype* top_mask = NULL;
    if (use_top_mask) {
      top_mask = top[1]->gpu_data();
    } else {
      mask = max_mask_.gpu_data();
    }
    // NOLINT_NEXT_LINE(whitespace/operators)
    SpatialWtaBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, mask,top_mask, bottom_diff);
    CUDA_POST_KERNEL_CHECK;
  }
}



// template <typename Dtype>
// __global__ void kernel_channel_max(const int num, const int channels,
//     const int height, const int width , const Dtype* data, Dtype* out) {
//   CUDA_KERNEL_LOOP(index, num * channels) {
//     const int c = index % channels;
//     const int n = index / channels;
//     Dtype maxval = -FLT_MAX;
//     for (int h = 0; h < height; ++h) {
//           for (int w = 0; w < width; ++w) {
//             maxval = max(data[h * width + w], maxval);
//           }
//     }
//     out[index] = maxval;
//   }
// }
//
// template <typename Dtype>
// __global__ void kernel_wta_forward(const int num, const int channels,
//     const int height, const int width , const Dtype* bottom_data, const Dtype* channel_max, Dtype* top_data) {
//   CUDA_KERNEL_LOOP(index, num) {
//     const int c = index / width / height % channels;
//     const int n = index / width / height/ channels;
//
//     top_data[index] = (bottom_data[index] == channel_max[n*c]) ? bottom_data[index] : 0;
//   }
// }
//
// template <typename Dtype>
// __global__ void kernel_wta_backward(const int num, const int channels,
//     const int height, const int width , const Dtype* bottom_data,
//     const Dtype* channel_max, const Dtype* const top_diff,
//     Dtype* bottom_diff) {
//   CUDA_KERNEL_LOOP(index, num) {
//     const int c = index / width / height % channels;
//     const int n = index / width / height/ channels;
//
//     bottom_diff[index] = (bottom_data[index] == channel_max[n*c]) ? top_diff[index] : 0;
//   }
// }
//
// (const int num, const int channels,
//     const int height, const int width , const Dtype* data, Dtype* out)
//
// template <typename Dtype>
// void SpatialConvWtaLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
//     const vector<Blob<Dtype>*>& top) {
//   const Dtype* bottom_data = bottom[0]->gpu_data();
//   Dtype* top_data = top[0]->mutable_gpu_data();
//   //const int count = bottom[0]->count();
//   const int num_data_slice = bottom[0]->num() * channels_;
//   // NOLINT_NEXT_LINE(whitespace/operators)
//   kernel_wta_forward<Dtype><<<CAFFE_GET_BLOCKS(num_data_slice), CAFFE_CUDA_NUM_THREADS>>>(
//     bottom[0]->num(), channels_, height_, width_, bottom_data, channel_max_data);
//   CUDA_POST_KERNEL_CHECK;
//   // << " count: " << count << " bottom_data: "
//   //     << (unsigned long)bottom_data
//   //     << " top_data: " << (unsigned long)top_data
//   //     << " blocks: " << CAFFE_GET_BLOCKS(count)
//   //     << " threads: " << CAFFE_CUDA_NUM_THREADS;
// }
// template <typename Dtype>
// void SpatialConvWtaLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
//     const vector<bool>& propagate_down,
//     const vector<Blob<Dtype>*>& bottom) {
//   if (propagate_down[0]) {
//     const Dtype* bottom_data = bottom[0]->gpu_data();
//     const Dtype* top_diff = top[0]->gpu_diff();
//     Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
//     const int count = bottom[0]->count();
//     const int num_data_slice = bottom[0]->num() * channels_;
//     // NOLINT_NEXT_LINE(whitespace/operators)
//     SpatialConvWtaBackward<Dtype><<<CAFFE_GET_BLOCKS(num_data_slice), CAFFE_CUDA_NUM_THREADS>>>(
//         num_data_slice, top_diff, bottom[0]->num(), channels_, height_, width_, bottom_diff, bottom_data);
//     CUDA_POST_KERNEL_CHECK;
//   }
// }



INSTANTIATE_LAYER_GPU_FUNCS(SpatialWtaLayer);


}  // namespace caffe
